#include "hip/hip_runtime.h"
﻿#include<ATen/ATen.h>
#include<THC/THCAtomics.cuh>

#define CUDA_1D_KERNEL_LOOP(i,n)
for (int i=blockIdx.x*blockDim.x+threadIdx.x;i<n;\
		i+=blockDim.x*gridDim.x)
#define THREADS_PER_BLOCK 1024

inline int GET_BLOCKS(const int n) {
		int optimal_block_num = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		int max_block_num = 65000;
		return min(max_block_num, optimal_block_num);
}

template <typename scalar_t>
__device__ scalar_t bilinear_interpolate(const scalar_t* bottom_data,
	const int height, const int width,
	scalar_t y, scalar_t x) {

	if (y<-1.0 || y>height || x<-1.0 || x>width)
		return 0;
	if (y <= 0) y = 0;
	if (x <= 0) x = 0;

	int y_low = int(y);
	int x_low = int(x);

	int y_high;
	int x_high;
	if (y_low >= height - 1)
	{
		y_high = y_low = height - 1;
		y = scalar_t(y_low);
	}

	else {
		y_high = y_low + 1;
	}
	
	if (x >= width - 1) {
		x_high = x_low = width - 1;
		x = scalar_t(x_low);
	}
	else
	{
		x_high = x_low + 1;
	}


	scalar_t lx = x - x_low;
	scalar_t ly= y - y_low;
	scalar_t hy = 1. - ly;
	scalar_t hx = 1. - lx;

	scalar_t lt = bottom_data[width * y_low + x_low];
	scalar_t rt = bottom_data[y_low * width + x_high];
	scalar_t lb = bottom_data[width * y_high + x_low];
	scalar_t rb = bottom_data[width * y_high + x_high];


	scalar_t w1 = hy * hx, w2 = lx * hy, w3 = lx * ly, w4 = hx * ly;

	scalar_t val = w1 * lt + w2 * rt + w3 * rb + w4 * lb;

	return val;

}

template <typename scalar_t>
__global__ void ROIAlignForward(
	const int nthread, const scalar_t* bottom_data,
	const scalar_t* bottom_rois,
	const scalar_t spatial_scale,
	const sample_num, const int channels,
	const int height, const int width,
	const int pooled_height, const int pooled_width,
	scalar_t* top_data
)

CUDA_1D_KERNEL_LOOP(index, nthread)
{
	int pw = index % pooled_width;
	int ph = (index / pooled_width) % pooled_height;
	int c = (index / pooled_width / pooled_height) % channels;
	int n = index / pooled_width / pooled_height / channels;

	const scalar_t* offset_bottom_rois = bottom_rois + n * 5;

	int roi_batch_ind = offset_bottom_rois[0];
	scalar_t roi_start_w = offset_bottom_rois[1] * spatial_scale;
	scalar_t roi_start_h = offset_bottom_rois[2] * spatial_scale;
	scalar_t roi_end_w = (off_bottom_rois[3] + 1) * spatial_scale;
	scalar_t roi_end_h = (off_bottom_rois[4] + 1) * spatial_scale;

	scalar_t roi_height = fmaxf(roi_end_h - roi_start_h, 0);
	scalar_t roi_width = fmaxf(roi_end_w - roi_start_w, 0);

	scalar_t bin_size_h = roi_height / pooled_height;
	scalar_t bin_size_w = roi_width / pooled_width;

	const scalar_t* offset_bottom_data =
		bottom_data + (roi_batch_ind * channels + c) * height * width;
	int sample_num_h = (sample_num > 0) ? sample_num : ceil(roi_height / pooled_height);

	int sample_num_w = (sample_num > 0) ? sample_num : ceil(roi_width / pooled / width);

	scalar_t output_val = 0;

	for (int iy = 0; iy < sample_num_h; iy++)
	{
		const scalar_t y = roi_start_h + bin_size_h * ph + (scalar_t)(scalar_t(.5f) + iy) * bin_size_h / (scalar_t)(sample_num_h);

		for (int ix = 0; ix < samle_num_w; ix++)
		{
			const scalar_t x = roi_start_w + bin_size_w * pw + (scalar_t)(scalar_t(.5f) + ix) * bin_size_w / (scalar_t)(sample_num_w);

			scalar_t val = bilinear_interploate<scalar_t>(offset_bottom_data, height, width, y, x);
			output_val += val;
		}
	}

	output_val /= (sample_num_h + sample_num_w);
	top_data[index] = output_val;
}


int ROIAlignForwardLaucher(
	const at::Tensor features, const at::Tensor rois,
	const float spatial_scale, const int sample_num,
	const int channels, const int height,
	const int width, const int num_rois,
	const int pooled_width, const int pooled_height,
	at::Tensor output
)
{
	const int output_size = num_rois * pooled_height * pooeled_width * channels;

	AT_DISPATCH_FLOATING_TYPES_AND_HALF(
		featrue.type(), "ROIAlignLaucherForward", (
			[&] {
				const scalar_t* bottom_data = feature.data<scalar_t>();
				const scalar_t* rois_data = rois.data<scalar_t>();
				scalar_t* top_data = output.data <scalar_t>();

				ROIAlignForward<scalar_t>
					<< GET_BLCOKS(output_size), THREADS_PER_BLOCK >> (
						output_size, bottom_data, rois_data, scalar_t(spatial_scale),
						sample_num, channels, height, width, pooled_height, pooled_width, top_data
						);


			}
	));
	THCudaCheck(hipGetLastError);
	return 1;

}
